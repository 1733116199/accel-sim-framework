#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cassert>
#include <string>

// Each thread block is maxed out (1024)
#define NT 1024

// Common macro for CUDA applications
#define gpuErrorCheck(ans, abort) \
  { gpuAssert((ans), __FILE__, __LINE__, abort); }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "assert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) {
      exit(code);
    }
  }
}

/**
 * This is a kernel that does "num_ops" MAC for every 2 memory accesses
 * Arithmetic intensity = num_ops (FLOPS per float accessed)
 */
__global__ void measure_max_flops(float *a, uint num_ops) {
  // uint i = blockIdx.x * NT + threadIdx.x;
  uint i;
  asm("mad.lo.u32 %0, %1, %2, %3;"
      : "=r"(i)
      : "r"(blockIdx.x), "n"(NT), "r"(threadIdx.x));

  // float * addr = &a[i];
  float *addr;
  asm("mad.wide.u32 %0, %1, %2, %3;"
      : "=l"(addr)
      : "r"(i), "n"(sizeof(float)), "l"(a));

  float input = *addr;
  float output = 0;

  for (int j = 0; j < num_ops; j++) {
    // Each iteration is a MAC op
    // Use inline assembly here to prevent compiler optimization
    // Equivalent C code: output += input * input;
    asm("fma.rn.f32 %0, %1, %1, %0;" : "+f"(output) : "f"(input));
  }
  *addr = output;
}

// common macro for CUDA kernels
#define ALLOC_FLOAT_GPU(SYMBOL, NUM, INIT_CODE)                             \
  float *SYMBOL;                                                            \
  float *SYMBOL##_device;                                                   \
  {                                                                         \
    SYMBOL = (float *)calloc(NUM, sizeof(float));                           \
    assert(SYMBOL != NULL);                                                 \
    INIT_CODE;                                                              \
    gpuErrorCheck(hipMalloc(&SYMBOL##_device, sizeof(float) * NUM), true); \
    gpuErrorCheck(hipMemcpy(SYMBOL##_device, SYMBOL, sizeof(float) * NUM,  \
                             hipMemcpyHostToDevice),                       \
                  true);                                                    \
  }

// common macro for CUDA kernels
#define TIME(CODE, SYMBOL)                      \
  float SYMBOL;                                 \
  {                                             \
    hipEvent_t start, stop;                    \
    hipEventCreate(&start);                    \
    hipEventCreate(&stop);                     \
    hipEventRecord(start);                     \
    CODE;                                       \
    hipDeviceSynchronize();                    \
    hipEventRecord(stop);                      \
    hipEventSynchronize(stop);                 \
    hipEventElapsedTime(&SYMBOL, start, stop); \
  }

int main(int argc, char **argv) {
  unsigned long long num_blocks;
  uint num_ops;
  if (argc >= 3) {
    num_blocks = std::stoull(argv[1]);
    num_ops = std::stoul(argv[2]);
  } else {
    num_blocks = 1048756ull;
    num_ops = 7000;
  }

  unsigned long long N = num_blocks * NT;

  ALLOC_FLOAT_GPU(a, N, {
    for (unsigned long long i = 0; i < N; i++) {
      a[i] = i;
    }
  });

  // launch and time the kernel
  TIME((measure_max_flops<<<num_blocks, NT>>>(a_device, num_ops)), ms);

  gpuErrorCheck(
      hipMemcpy(a, a_device, sizeof(float) * N, hipMemcpyDeviceToHost), true);

  for (int i = 0; i < 20; i++) {
    // printf("Actual: %.0f Expected: %.0f\n", a[i], (float)i * i * NUM_OPS);
    if (i < 20) {
      assert((i * i * num_ops) == (unsigned long long)a[i]);
    }
  }

  double gflops = num_blocks * NT * num_ops * 2 / (ms / 1000.0) / 1000000000.0;
  double ai = num_ops / 2.0 / sizeof(float);
  double bw = num_blocks * NT * 2 * sizeof(float) / (ms / 1000.0) / 1000000000;

  printf("Time Elapsed: %.4f ms\n", ms);
  printf("Arithmetic Intensity (flops per byte): %.4f\n", ai);
  printf("Effective bandwidth (GB per second): %.4f\n", bw);
  printf("GFLOPS Measured: %.2f\n", gflops);
}