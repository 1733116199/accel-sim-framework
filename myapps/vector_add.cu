#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024

__global__ void vector_add(float *a, float *b1, float * b2, float *c, int size) {
  int i = threadIdx.x;
  if (i < size) {
    if((i % 2) == 0)
        c[i] = a[i] + b1[i];
    else
        c[i] = a[i] + b2[i];
  }
}

int main() {
  float a[N];
  float b1[N];
  float b2[N];
  float c[N];
  for (int i = 0; i < N; i++) {
    a[i] = i;
    b1[i] = 2 * i;
    b2[i] = 3 * i;
  }
  float *a_device, *b1_device, *b2_device, *c_device;
  hipMalloc(&a_device, sizeof(a));
  hipMalloc(&b1_device, sizeof(b1));
  hipMalloc(&b2_device, sizeof(b2));
  hipMalloc(&c_device, sizeof(c));

  hipMemcpy(a_device, a, sizeof(a), hipMemcpyHostToDevice);
  hipMemcpy(b1_device, b1, sizeof(b1), hipMemcpyHostToDevice);
  hipMemcpy(b2_device, b2, sizeof(b2), hipMemcpyHostToDevice);

  vector_add<<<1, N>>>(a_device, b1_device, b2_device,c_device, N);
  hipMemcpy(c, c_device, sizeof(c), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++) {
    printf("%.2f\n", c[i]);
  }
}