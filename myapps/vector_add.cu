#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024

__global__ void vector_add(float *a, float *b, float *c, int size) {
  int i = threadIdx.x;
  if (i < size) {
    c[i] = a[i] + b[i];
  }
}

int main() {
  float a[N];
  float b[N];
  float c[N];
  for (int i = 0; i < N; i++) {
    a[i] = i;
    b[i] = 2 * i;
  }
  float *a_device, *b_device, *c_device;
  hipMalloc(&a_device, sizeof(a));
  hipMalloc(&b_device, sizeof(b));
  hipMalloc(&c_device, sizeof(c));

  hipMemcpy(a_device, a, sizeof(a), hipMemcpyHostToDevice);
  hipMemcpy(b_device, b, sizeof(b), hipMemcpyHostToDevice);

  vector_add<<<1, N>>>(a_device, b_device, c_device, N);
  hipMemcpy(c, c_device, sizeof(c), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++) {
    printf("%.2f\n", c[i]);
  }
}